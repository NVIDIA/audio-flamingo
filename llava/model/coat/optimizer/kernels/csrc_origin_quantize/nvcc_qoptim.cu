#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <hip/hip_fp8.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>

#include <fstream>
#include <iomanip>
#include <iostream>

namespace cg = cooperative_groups;
#define WARPSIZE 32
#define QGROUPSIZE 128
#define QUANT_MIN_VAL 1e-20

template <typename T>
inline float fp8_dtype_max(const T &variable) {
  if (std::is_same<T, __hip_fp8_e4m3_fnuz>::value) {
    return 448;
  } else if (std::is_same<T, __hip_fp8_e5m2_fnuz>::value) {
    return 57344;
  } else {
    throw "Unsupported data format";
  }
}

typedef enum { fp8_adamw } myCsrcKernels;

void fp8_adamw_cpu(float *params, float *grads, float *fp_exp_avg,
                   float *fp_exp_avg_sq, float beta1, float beta2, float lr,
                   float wd, float eps, int step, int qgroup_size, int M,
                   int N) {
  for (int idx = 0; idx < M * N; idx++) {
    fp_exp_avg[idx] = beta1 * fp_exp_avg[idx] + (1 - beta1) * grads[idx];
    fp_exp_avg_sq[idx] =
        beta2 * fp_exp_avg_sq[idx] + (1 - beta2) * grads[idx] * grads[idx];

    const float correction1 = 1.0f - powf(beta1, step);
    const float correction2_sqrt = sqrtf(1.0f - powf(beta2, step));

    float denom =
        (sqrtf(fp_exp_avg_sq[idx]) / correction2_sqrt + eps) * correction1;
    float update = (fp_exp_avg[idx] / denom) + (wd * params[idx]);
    params[idx] = params[idx] - (lr * update);
  }
}

template <typename T>
void printFloatArrayToFile(T *array, int M, int N,
                           const std::string &outputFileName) {
  std::ofstream outputFile(outputFileName);
  if (!outputFile.is_open()) {
    std::cout << "Failed to open the file." << std::endl;
    return;
  }

  for (int i = 0; i < M; i++) {
    for (int j = 0; j < N; j++) {
      int index = i * N + j;
      outputFile << std::setw(10) << std::right << std::fixed
                 << std::setprecision(6) << (float)array[index] << " ";
      if (j == N - 1) {
        outputFile << "\n";
      }
    }
  }
}

template <typename scalar_t>
__global__ void fp8_adamw_csrc(scalar_t *__restrict__ params,
                               scalar_t *__restrict__ grads,
                               __hip_fp8_e4m3_fnuz *__restrict__ exp_avg,
                               float *__restrict__ scale_exp_avg,
                               __hip_fp8_e4m3_fnuz *__restrict__ exp_avg_sq,
                               float *__restrict__ scale_exp_avg_sq,
                               float beta1, float beta2, float lr, float wd,
                               float eps, int step, int qgroup_size,
                               int total_elements, int total_scale_elements) {
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const int scale_idx = blockIdx.x;

  float float_exp_avg, float_exp_avg_sq;
  float correction1, correction2_sqrt;
  float denom, update;

  if (idx < total_elements) {
    // dequantize the optimizer states
    float_exp_avg = float(exp_avg[idx]) * scale_exp_avg[scale_idx];
    float_exp_avg_sq = float(exp_avg_sq[idx]) * scale_exp_avg_sq[scale_idx];

    // calculation of optimizer.step()
    float_exp_avg = beta1 * float_exp_avg + (1 - beta1) * grads[idx];
    float_exp_avg_sq =
        beta2 * float_exp_avg_sq + (1 - beta2) * grads[idx] * grads[idx];

    correction1 = 1.0f - powf(beta1, step);
    correction2_sqrt = sqrtf(1.0f - powf(beta2, step));

    denom = (sqrtf(float_exp_avg_sq) / correction2_sqrt + eps) * correction1;
    update = (float_exp_avg / denom) + (wd * params[idx]);

    params[idx] = params[idx] - (lr * update);
  } else {
    float_exp_avg = 0.0f;
    float_exp_avg_sq = 0.0f;
  }

  //// quantize the first-order and second-order momentum
  int wid = threadIdx.x / WARPSIZE;

  // reduction within a warp

  __shared__ float sharedFirstMaxVal[32];
  __shared__ float sharedSecondMaxVal[32];
  cg::thread_block_tile<32> warpTile =
      cg::tiled_partition<32>(cg::this_thread_block());
  float firstMaxVal = fabsf(float_exp_avg);
  float secondMaxVal = fabsf(float_exp_avg_sq);

  for (int i = warpTile.size() / 2; i > 0; i /= 2) {
    float reduceFirstMaxVal = warpTile.shfl_down(firstMaxVal, i);
    float reduceSecondMaxVal = warpTile.shfl_down(secondMaxVal, i);
    firstMaxVal = fmax(firstMaxVal, fabsf(reduceFirstMaxVal));
    secondMaxVal = fmax(secondMaxVal, fabsf(reduceSecondMaxVal));
    // printf("First Max: %f\n", reduceFirstMaxVal);
  }
  int lane = warpTile.thread_rank();
  if (lane == 0) sharedFirstMaxVal[wid] = firstMaxVal;
  if (lane == 0) sharedSecondMaxVal[wid] = secondMaxVal;

  __syncthreads();

  // reduction within a block
  __shared__ float shared_absmax_exp_avg;
  __shared__ float shared_absmax_exp_avg_sq;
  firstMaxVal =
      (threadIdx.x < blockDim.x / warpSize) ? sharedFirstMaxVal[lane] : 0;
  secondMaxVal =
      (threadIdx.x < blockDim.x / warpSize) ? sharedSecondMaxVal[lane] : 0;
  if (wid == 0) {
    for (int offset = WARPSIZE / 2; offset > 0; offset /= 2) {
      float reduceFirstMaxVal =
          __shfl_down_sync(0xFFFFFFFF, firstMaxVal, offset);
      float reduceSecondMaxVal =
          __shfl_down_sync(0xFFFFFFFF, secondMaxVal, offset);
      firstMaxVal = fmax(firstMaxVal, fabsf(reduceFirstMaxVal));
      secondMaxVal = fmax(secondMaxVal, fabsf(reduceSecondMaxVal));
    }
    if (lane == 0) shared_absmax_exp_avg = firstMaxVal;
    if (lane == 0) shared_absmax_exp_avg_sq = secondMaxVal;
  }

  __syncthreads();

  if (idx < total_elements) {
    // float fp8MaxVal = fp8_dtype_max<__hip_fp8_e4m3_fnuz>(exp_avg[idx]);
    float fp8MaxVal = 448;

    shared_absmax_exp_avg = shared_absmax_exp_avg + QUANT_MIN_VAL;
    shared_absmax_exp_avg_sq = shared_absmax_exp_avg_sq + QUANT_MIN_VAL;

    float new_scale_exp_avg = shared_absmax_exp_avg / fp8MaxVal;
    float new_scale_exp_avg_sq = shared_absmax_exp_avg_sq / fp8MaxVal;

    // quantize the optimizer states
    __hip_fp8_e4m3_fnuz exp_avg_new =
        static_cast<__hip_fp8_e4m3_fnuz>(float_exp_avg / new_scale_exp_avg);
    __hip_fp8_e4m3_fnuz exp_avg_sq_new =
        static_cast<__hip_fp8_e4m3_fnuz>(float_exp_avg_sq / new_scale_exp_avg_sq);
    // __half exp_avg_new = static_cast<__half>(float_exp_avg /
    // new_scale_exp_avg);
    // __half exp_avg_sq_new = static_cast<__half>(float_exp_avg_sq /
    // new_scale_exp_avg_sq);

    // printf("idx: %d, float: %f, quantize: %f\n", idx, float_exp_avg,
    // (float)exp_avg_new * new_scale_exp_avg);

    // store the output
    exp_avg[idx] = exp_avg_new;
    exp_avg_sq[idx] = exp_avg_sq_new;
    scale_exp_avg[scale_idx] = new_scale_exp_avg;
    scale_exp_avg_sq[scale_idx] = new_scale_exp_avg_sq;
  }
}

template <myCsrcKernels algo>
void myKernelLauncher(float *params, float *grads, __hip_fp8_e4m3_fnuz *exp_avg,
                      float *scale_exp_avg, __hip_fp8_e4m3_fnuz *exp_avg_sq,
                      float *scale_exp_avg_sq, float beta1, float beta2,
                      float lr, float wd, float eps, int step, int qgroup_size,
                      int M, int N) {
  if (algo == fp8_adamw) {
    const int block_dim = 128;
    int grid_dim = (M * N + qgroup_size - 1) / block_dim;
    const dim3 gridDim(grid_dim);
    const dim3 blockDim(block_dim);
    printf("Yes!\n");
    fp8_adamw_csrc<float><<<gridDim, blockDim>>>(
        params, grads, exp_avg, scale_exp_avg, exp_avg_sq, scale_exp_avg_sq,
        beta1, beta2, lr, wd, eps, step, qgroup_size, M * N,
        int(floor(M * N / 128.)));
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
      std::cout << "CUDA error occurred in kernel launch: "
                << hipGetErrorString(error) << std::endl;
      return;
    }
    printf("Finish!\n");
  }
}

float testMaxError(void (*myGPUKernel)(float *, float *, __hip_fp8_e4m3_fnuz *,
                                       float *, __hip_fp8_e4m3_fnuz *, float *, float,
                                       float, float, float, float, int, int,
                                       int, int),
                   int M, int N) {
  size_t size_param = M * N * sizeof(float);
  size_t size_optim = M * N * sizeof(__hip_fp8_e4m3_fnuz);
  size_t size_scale = int(ceil(M * N / 128.)) * sizeof(float);

  // host tensor
  float *h_p, *h_g;
  __hip_fp8_e4m3_fnuz *h_m, *h_v;
  float *h_sm, *h_sv;
  float *h_fp_m, *h_fp_v;

  // device tensor
  float *d_p, *d_g;
  __hip_fp8_e4m3_fnuz *d_m, *d_v;
  float *d_sm, *d_sv;

  // device tensor transfer to host
  float *hd_p, *hd_g;
  __hip_fp8_e4m3_fnuz *hd_m, *hd_v;
  float *hd_sm, *hd_sv;
  float *hd_fp_m, *hd_fp_v;

  h_p = (float *)malloc(size_param);
  h_g = (float *)malloc(size_param);
  h_m = (__hip_fp8_e4m3_fnuz *)malloc(size_optim);
  h_v = (__hip_fp8_e4m3_fnuz *)malloc(size_optim);
  h_sm = (float *)malloc(size_scale);
  h_sv = (float *)malloc(size_scale);
  h_fp_m = (float *)malloc(size_param);
  h_fp_v = (float *)malloc(size_param);
  hipMalloc(&d_p, size_param);
  hipMalloc(&d_g, size_param);
  hipMalloc(&d_m, size_optim);
  hipMalloc(&d_v, size_optim);
  hipMalloc(&d_sm, size_scale);
  hipMalloc(&d_sv, size_scale);
  hd_p = (float *)malloc(size_param);
  hd_g = (float *)malloc(size_param);
  hd_m = (__hip_fp8_e4m3_fnuz *)malloc(size_optim);
  hd_v = (__hip_fp8_e4m3_fnuz *)malloc(size_optim);
  hd_sm = (float *)malloc(size_scale);
  hd_sv = (float *)malloc(size_scale);
  hd_fp_m = (float *)malloc(size_param);
  hd_fp_v = (float *)malloc(size_param);

  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    std::cout << "CUDA error occurred in data copy: "
              << hipGetErrorString(error) << std::endl;
    return 0.;
  }

  srand(0);
  // random initialization for CPU tensor
  for (int i = 0; i < M * N; i++) {
    h_p[i] = (float)(rand() / (float(RAND_MAX) / 10));
    h_g[i] = (float)(rand() / (float(RAND_MAX) / 10));
    h_m[i] = (__hip_fp8_e4m3_fnuz)(rand() / (float(RAND_MAX) / 10));
    h_v[i] = (__hip_fp8_e4m3_fnuz)(rand() / (float(RAND_MAX) / 10));
  }
  for (int i = 0; i < int(ceilf(M * N / 128.)); i++) {
    h_sm[i] = (float)(rand() / (float(RAND_MAX) / 10));
    h_sv[i] = (float)(rand() / (float(RAND_MAX) / 10));
    printf("scale is %f\n", h_sm[i]);
  }
  for (int i = 0; i < M * N; i++) {
    h_fp_m[i] = (float)h_m[i] * h_sm[int(floor(i / 128.))];
    h_fp_v[i] = (float)h_v[i] * h_sv[int(floor(i / 128.))];
  }
  float beta1 = 0.9, beta2 = 0.95, lr = 4e-4, wd = 0.1, eps = 1e-8;
  int step = 100, qgroup_size = 128;

  printFloatArrayToFile(h_p, M, N, "Past_CPU_param.txt");
  printFloatArrayToFile(h_g, M, N, "Past_CPU_grad.txt");
  printFloatArrayToFile(h_m, M, N, "Past_CPU_m1.txt");
  printFloatArrayToFile(h_sm, 1, int(ceilf(M * N / 128.)), "Past_CPU_ms.txt");
  printFloatArrayToFile(h_fp_m, M, N, "Past_CPU_mf.txt");
  printFloatArrayToFile(h_v, M, N, "Past_CPU_v2.txt");
  printFloatArrayToFile(h_sv, 1, int(ceilf(M * N / 128.)), "Past_CPU_vs.txt");
  printFloatArrayToFile(h_fp_v, M, N, "Past_CPU_vf.txt");

  hipMemcpy(d_p, h_p, size_param, hipMemcpyHostToDevice);
  hipMemcpy(d_g, h_g, size_param, hipMemcpyHostToDevice);
  hipMemcpy(d_m, h_m, size_optim, hipMemcpyHostToDevice);
  hipMemcpy(d_v, h_v, size_optim, hipMemcpyHostToDevice);
  hipMemcpy(d_sm, h_sm, size_scale, hipMemcpyHostToDevice);
  hipMemcpy(d_sv, h_sv, size_scale, hipMemcpyHostToDevice);

  fp8_adamw_cpu(h_p, h_g, h_fp_m, h_fp_v, beta1, beta2, lr, wd, eps, step,
                qgroup_size, M, N);

  if (error != hipSuccess) {
    std::cout << "CUDA error occurred in data initialization: "
              << hipGetErrorString(error) << std::endl;
    return 0.;
  }

  myGPUKernel(d_p, d_g, d_m, d_sm, d_v, d_sv, beta1, beta2, lr, wd, eps, step,
              qgroup_size, M, N);

  hipMemcpy(hd_p, d_p, size_param, hipMemcpyDeviceToHost);
  hipMemcpy(hd_g, d_g, size_param, hipMemcpyDeviceToHost);
  hipMemcpy(hd_m, d_m, size_optim, hipMemcpyDeviceToHost);
  hipMemcpy(hd_v, d_v, size_optim, hipMemcpyDeviceToHost);
  hipMemcpy(hd_sm, d_sm, size_scale, hipMemcpyDeviceToHost);
  hipMemcpy(hd_sv, d_sv, size_scale, hipMemcpyDeviceToHost);

  for (int i = 0; i < M * N; i++) {
    hd_fp_m[i] = (float)hd_m[i] * hd_sm[int(floor(i / 128.))];
    hd_fp_v[i] = (float)hd_v[i] * hd_sv[int(floor(i / 128.))];
  }
  printFloatArrayToFile(h_p, M, N, "CPU_param.txt");
  printFloatArrayToFile(hd_p, M, N, "GPU_param.txt");
  printFloatArrayToFile(h_g, M, N, "CPU_grad.txt");
  printFloatArrayToFile(hd_g, M, N, "GPU_grad.txt");
  printFloatArrayToFile(h_m, M, N, "CPU_m1.txt");
  printFloatArrayToFile(h_sm, 1, int(ceilf(M * N / 128.)), "CPU_ms.txt");
  printFloatArrayToFile(h_fp_m, M, N, "CPU_mf.txt");
  printFloatArrayToFile(hd_m, M, N, "GPU_m1.txt");
  printFloatArrayToFile(hd_sm, 1, int(ceilf(M * N / 128.)), "GPU_ms.txt");
  printFloatArrayToFile(hd_fp_m, M, N, "GPU_mf.txt");
  printFloatArrayToFile(h_v, M, N, "CPU_v2.txt");
  printFloatArrayToFile(h_sv, 1, int(ceilf(M * N / 128.)), "CPU_vs.txt");
  printFloatArrayToFile(h_fp_v, M, N, "CPU_vf.txt");
  printFloatArrayToFile(hd_v, M, N, "GPU_v2.txt");
  printFloatArrayToFile(hd_sv, 1, int(ceilf(M * N / 128.)), "GPU_vs.txt");
  printFloatArrayToFile(hd_fp_v, M, N, "GPU_vf.txt");

  return 0.;
}

int main() {
  const int M = 1, N = 7;
  float max_error = testMaxError(myKernelLauncher<fp8_adamw>, M, N);
}
