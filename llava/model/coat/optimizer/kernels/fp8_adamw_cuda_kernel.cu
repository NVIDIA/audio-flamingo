#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <hip/hip_fp8.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <torch/extension.h>

#define QUANT_MIN_VAL 1e-20

namespace cg = cooperative_groups;
#define WARPSIZE 32

template <typename scalar_t>
__global__ void fp8_adamw_cuda_kernel(
    scalar_t* __restrict__ params, scalar_t* __restrict__ grads,
    __hip_fp8_e4m3_fnuz* __restrict__ exp_avg, float* __restrict__ scale_exp_avg,
    __hip_fp8_e4m3_fnuz* __restrict__ exp_avg_sq,
    float* __restrict__ scale_exp_avg_sq, float beta1, float beta2, float lr,
    float wd, float eps, int step, int qgroup_size, int total_elements,
    int total_scale_elements) {
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const int scale_idx = blockIdx.x;

  float float_exp_avg, float_exp_avg_sq;
  float correction1, correction2_sqrt;
  float denom, update;

  if (idx < total_elements) {
    // dequantize the optimizer states
    float_exp_avg = float(exp_avg[idx]) * scale_exp_avg[scale_idx];
    float_exp_avg_sq = float(exp_avg_sq[idx]) * scale_exp_avg_sq[scale_idx];

    // calculation of optimizer.step()
    float_exp_avg = beta1 * float_exp_avg + (1 - beta1) * grads[idx];
    float_exp_avg_sq =
        beta2 * float_exp_avg_sq + (1 - beta2) * grads[idx] * grads[idx];

    correction1 = 1.0f - powf(beta1, step);
    correction2_sqrt = sqrtf(1.0f - powf(beta2, step));

    denom = (sqrtf(float_exp_avg_sq) / correction2_sqrt + eps) * correction1;
    update = (float_exp_avg / denom) + (wd * params[idx]);

    params[idx] = params[idx] - (lr * update);
  } else {
    float_exp_avg = 0.0f;
    float_exp_avg_sq = 0.0f;
  }

  //// quantize the first-order and second-order momentum
  int wid = threadIdx.x / WARPSIZE;

  // reduction within a warp

  __shared__ float sharedFirstMaxVal[32];
  __shared__ float sharedSecondMaxVal[32];
  cg::thread_block_tile<32> warpTile =
      cg::tiled_partition<32>(cg::this_thread_block());
  float firstMaxVal = fabsf(float_exp_avg);
  float secondMaxVal = fabsf(float_exp_avg_sq);

  for (int i = warpTile.size() / 2; i > 0; i /= 2) {
    float reduceFirstMaxVal = warpTile.shfl_down(firstMaxVal, i);
    float reduceSecondMaxVal = warpTile.shfl_down(secondMaxVal, i);
    firstMaxVal = fmax(firstMaxVal, fabsf(reduceFirstMaxVal));
    secondMaxVal = fmax(secondMaxVal, fabsf(reduceSecondMaxVal));
    // printf("First Max: %f\n", reduceFirstMaxVal);
  }
  int lane = warpTile.thread_rank();
  if (lane == 0) sharedFirstMaxVal[wid] = firstMaxVal;
  if (lane == 0) sharedSecondMaxVal[wid] = secondMaxVal;

  __syncthreads();

  // reduction within a block
  __shared__ float shared_absmax_exp_avg;
  __shared__ float shared_absmax_exp_avg_sq;
  firstMaxVal =
      (threadIdx.x < blockDim.x / warpSize) ? sharedFirstMaxVal[lane] : 0;
  secondMaxVal =
      (threadIdx.x < blockDim.x / warpSize) ? sharedSecondMaxVal[lane] : 0;
  if (wid == 0) {
    for (int offset = WARPSIZE / 2; offset > 0; offset /= 2) {
      float reduceFirstMaxVal =
          __shfl_down_sync(0xFFFFFFFF, firstMaxVal, offset);
      float reduceSecondMaxVal =
          __shfl_down_sync(0xFFFFFFFF, secondMaxVal, offset);
      firstMaxVal = fmax(firstMaxVal, fabsf(reduceFirstMaxVal));
      secondMaxVal = fmax(secondMaxVal, fabsf(reduceSecondMaxVal));
    }
    if (lane == 0) shared_absmax_exp_avg = firstMaxVal;
    if (lane == 0) shared_absmax_exp_avg_sq = secondMaxVal;
  }

  __syncthreads();

  if (idx < total_elements) {
    // float fp8MaxVal = fp8_dtype_max<__hip_fp8_e4m3_fnuz>(exp_avg[idx]);
    float fp8MaxVal = 448;

    shared_absmax_exp_avg = shared_absmax_exp_avg + QUANT_MIN_VAL;
    shared_absmax_exp_avg_sq = shared_absmax_exp_avg_sq + QUANT_MIN_VAL;

    float new_scale_exp_avg = shared_absmax_exp_avg / fp8MaxVal;
    float new_scale_exp_avg_sq = shared_absmax_exp_avg_sq / fp8MaxVal;

    // quantize the optimizer states
    __hip_fp8_e4m3_fnuz exp_avg_new =
        static_cast<__hip_fp8_e4m3_fnuz>(float_exp_avg / new_scale_exp_avg);
    __hip_fp8_e4m3_fnuz exp_avg_sq_new =
        static_cast<__hip_fp8_e4m3_fnuz>(float_exp_avg_sq / new_scale_exp_avg_sq);
    // __half exp_avg_new = static_cast<__half>(float_exp_avg /
    // new_scale_exp_avg);
    // __half exp_avg_sq_new = static_cast<__half>(float_exp_avg_sq /
    // new_scale_exp_avg_sq);

    // printf("idx: %d, float: %f, quantize: %f\n", idx, float_exp_avg,
    // (float)exp_avg_new * new_scale_exp_avg);

    // store the output
    exp_avg[idx] = exp_avg_new;
    exp_avg_sq[idx] = exp_avg_sq_new;
    scale_exp_avg[scale_idx] = new_scale_exp_avg;
    scale_exp_avg_sq[scale_idx] = new_scale_exp_avg_sq;
  }
}

void FP8_AdamW_cuda(torch::Tensor params,   // parameter
                    torch::Tensor grads,    // gradient
                    torch::Tensor exp_avg,  // first order momentum
                    torch::Tensor scale_exp_avg,
                    torch::Tensor exp_avg_sq,  // second order momentum
                    torch::Tensor scale_exp_avg_sq, float beta1, float beta2,
                    float lr, float wd, float eps, int step,
                    int qgroup_size) {  // other parameters

  // CUDA Blocks
  int total_elements = params.numel();
  int total_scale_elements = scale_exp_avg.numel();
  AT_ASSERTM(qgroup_size == 128,
             "Only Support 128 per-group quantization currently");
  const int block_dim = 128;  // This should equal to the qgroup_size
  int grid_dim = (total_elements + qgroup_size - 1) / block_dim;
  AT_ASSERTM(grid_dim == scale_exp_avg.numel());
  AT_ASSERTM(grid_dim == scale_exp_avg_sq.numel());
  const dim3 blocks(grid_dim);

  // Execution
  AT_DISPATCH_FLOATING_TYPES_AND2(
      at::kBFloat16, at::kHalf, params.scalar_type(), "fp8_adamw", ([&] {
        fp8_adamw_cuda_kernel<scalar_t><<<blocks, block_dim>>>(
            params.data_ptr<scalar_t>(), grads.data_ptr<scalar_t>(),
            (__hip_fp8_e4m3_fnuz*)exp_avg.data_ptr<at::Float8_e4m3fn>(),
            scale_exp_avg.data_ptr<float>(),
            (__hip_fp8_e4m3_fnuz*)exp_avg_sq.data_ptr<at::Float8_e4m3fn>(),
            scale_exp_avg_sq.data_ptr<float>(), beta1, beta2, lr, wd, eps, step,
            qgroup_size, total_elements, total_scale_elements);
      }));
}
